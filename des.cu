#include "hip/hip_runtime.h"
// -----------------------------------------------------------------
// descracker - brute forcer for legacy Unix DES based password hash
// Copyright (C) 2008-2024  Gabriele Bonacini
//
// This program is free software; you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation; either version 3 of the License, or
// (at your option) any later version.
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software Foundation,
// Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301  USA
// 
// CREDITS: DES impementation extracted from OpenSSL library 
//          All credits to original authors.
// -----------------------------------------------------------------

#include <iostream>
#include <fstream>
#include <algorithm>
#include <filesystem>

#include <cuda/semaphore>

#include "descuda.hpp"

namespace descrack {

using std::cout,
      std::cerr,
      std::string,
      std::abort,
      std::fill_n,
      std::copy_n,
      std::ifstream,
      std::filesystem::is_regular_file,
      cuda::binary_semaphore;

using DES_LONG=unsigned int;
using DES_cblock=unsigned char;

struct DES_cuda_ks {
    union {
        DES_cblock cblock[8]; 
        DES_LONG deslong[2];
    } ks[16];
};
using DES_key_schedule_cuda=DES_cuda_ks;

#define l2c(l,c)       (*((c)++)=(unsigned char)(((l)     )&0xff), \
                        *((c)++)=(unsigned char)(((l)>> 8L)&0xff), \
                        *((c)++)=(unsigned char)(((l)>>16L)&0xff), \
                        *((c)++)=(unsigned char)(((l)>>24L)&0xff))

#define D_ENCRYPT(LL,R,S) { \
        LOAD_DATA_tmp(R,S,u,t,E0,E1); \
        t=ROTATE(t,4); \
        LL^= \
            DES_SPtrans[0][(u>> 2L)&0x3f]^ \
            DES_SPtrans[2][(u>>10L)&0x3f]^ \
            DES_SPtrans[4][(u>>18L)&0x3f]^ \
            DES_SPtrans[6][(u>>26L)&0x3f]^ \
            DES_SPtrans[1][(t>> 2L)&0x3f]^ \
            DES_SPtrans[3][(t>>10L)&0x3f]^ \
            DES_SPtrans[5][(t>>18L)&0x3f]^ \
            DES_SPtrans[7][(t>>26L)&0x3f]; }

#define ROTATE(a,n)     (((a)>>(n))+((a)<<(32-(n))))

#define PERM_OP(a,b,t,n,m) ((t)=((((a)>>(n))^(b))&(m)),\
        (b)^=(t),\
        (a)^=((t)<<(n)))

#define LOAD_DATA_tmp(R,S,u,t,E0,E1) \
        { DES_LONG tmp; LOAD_DATA(R,S,u,t,E0,E1,tmp); }

#define LOAD_DATA(R,S,u,t,E0,E1,tmp) \
        t=R^(R>>16L); \
        u=t&E0; t&=E1; \
        tmp=(u<<16); u^=R^s[S  ]; u^=tmp; \
        tmp=(t<<16); t^=R^s[S+1]; t^=tmp

#define HPERM_OP(a,t,n,m) ((t)=((((a)<<(16-(n)))^(a))&(m)),\
        (a)=(a)^(t)^(t>>(16-(n))))

#define c2l(c,l)       (l =((DES_LONG)(*((c)++)))     , \
                        l|=((DES_LONG)(*((c)++)))<< 8L, \
                        l|=((DES_LONG)(*((c)++)))<<16L, \
                        l|=((DES_LONG)(*((c)++)))<<24L)

__device__ void DES_hash_cuda(const char *buf, const char *salt, char *ret) { // ret point to a char[14]
    unsigned int          x, y;
    DES_LONG              Eswap0, Eswap1, out[2], ll;
    DES_cblock            key[8];
    DES_key_schedule_cuda ks;
    unsigned char         bb[9], *b = bb, c, u;

    #include "desdata.h"

    x = ret[0] = salt[0];
    if (x == 0 || x >= sizeof(con_salt))
        return;
    Eswap0 = con_salt[x] << 2;
    x = ret[1] = salt[1];
    if (x == 0 || x >= sizeof(con_salt))
        return;
    Eswap1 = con_salt[x] << 6;

    key[0] = 0; key[1] = 0; key[2] = 0; key[3] = 0; key[4] = 0; key[5] = 0; key[6] = 0; key[7] = 0;
    int calc = 1;
    if(calc){ c = *(buf++); if(c){ key[0] = (c << 1); } else { calc = 0;}; }
    if(calc){ c = *(buf++); if(c){ key[1] = (c << 1); } else { calc = 0;}; }
    if(calc){ c = *(buf++); if(c){ key[2] = (c << 1); } else { calc = 0;}; }
    if(calc){ c = *(buf++); if(c){ key[3] = (c << 1); } else { calc = 0;}; }
    if(calc){ c = *(buf++); if(c){ key[4] = (c << 1); } else { calc = 0;}; }
    if(calc){ c = *(buf++); if(c){ key[5] = (c << 1); } else { calc = 0;}; }
    if(calc){ c = *(buf++); if(c){ key[6] = (c << 1); } else { calc = 0;}; }
    if(calc){ c = *(buf++); if(c){ key[7] = (c << 1); } else { calc = 0;}; }

    static const int    shifts2[16] = { 0, 0, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 0 };
    DES_LONG            cc, d, t, s, t2, *k = (DES_LONG*)&ks.ks[0];
    const unsigned char *in = &key[0];

    c2l(in, cc);
    c2l(in, d);

    PERM_OP(d, cc, t, 4, 0x0f0f0f0fL);
    HPERM_OP(cc, t, -2, 0xcccc0000L);
    HPERM_OP(d, t, -2, 0xcccc0000L);
    PERM_OP(d, cc, t, 1, 0x55555555L);
    PERM_OP(cc, d, t, 8, 0x00ff00ffL);
    PERM_OP(d, cc, t, 1, 0x55555555L);
    d = (((d & 0x000000ffL) << 16L) | (d & 0x0000ff00L) |
         ((d & 0x00ff0000L) >> 16L) | ((cc & 0xf0000000L) >> 4L));
    cc &= 0x0fffffffL;

    /* 1 */ if(shifts2[0]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL;
    *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L));
    *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 2 */ if(shifts2[1]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL;
    *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L));
    *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 3 */ if(shifts2[2]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL;
    *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L));
    *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 4 */ if(shifts2[3]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 5 */ if(shifts2[4]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 6 */ if(shifts2[5]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 7 */ if(shifts2[6]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 8 */ if(shifts2[7]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 9 */ if(shifts2[8]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 10 */ if(shifts2[9]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else { cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 11 */ if(shifts2[10]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else{ cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 12 */ if(shifts2[11]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else{ cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 13 */ if(shifts2[12]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else{ cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 14 */ if(shifts2[13]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else{ cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 15 */ if(shifts2[14]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else{ cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;
    /* 16 */ if(shifts2[15]){ cc = ((cc >> 2L) | (cc << 26L)); d = ((d >> 2L) | (d << 26L)); } else{ cc = ((cc >> 1L) | (cc << 27L)); d = ((d >> 1L) | (d << 27L));}
    cc &= 0x0fffffffL;
    d &= 0x0fffffffL;
    s = des_skb[0][(cc) & 0x3f] | des_skb[1][((cc >> 6L) & 0x03) | ((cc >> 7L) & 0x3c)] | des_skb[2][((cc >> 13L) & 0x0f) | 
	((cc >> 14L) & 0x30)] | des_skb[3][((cc >> 20L) & 0x01) | ((cc >> 21L) & 0x06) | ((cc >> 22L) & 0x38)];
    t = des_skb[4][(d) & 0x3f] | des_skb[5][((d >> 7L) & 0x03) | ((d >> 8L) & 0x3c)] | des_skb[6][(d >> 15L) & 0x3f] | 
	des_skb[7][((d >> 21L) & 0x0f) | ((d >> 22L) & 0x30)];
    t2 = ((t << 16L) | (s & 0x0000ffffL)) & 0xffffffffL; *(k++) = ROTATE(t2, 30) & 0xffffffffL;
    t2 = ((s >> 16L) | (t & 0xffff0000L)); *(k++) = ROTATE(t2, 26) & 0xffffffffL;

  { 
        DES_LONG l = 0, r = 0, t, u, *s = (DES_LONG *)&ks, E0 = Eswap0, E1 = Eswap1;

        /* 1 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 2 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 3 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 4 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 5 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 6 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 7 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 8 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 9 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 10 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 11 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 12 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 13 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 14 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 15 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 16 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 17 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 18 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 19 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 20 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 21 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 22 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 23 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 24 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;
        /* 25 */ D_ENCRYPT(l, r, 0);  D_ENCRYPT(r, l, 2); D_ENCRYPT(l, r, 4); D_ENCRYPT(r, l, 6); D_ENCRYPT(l, r, 8); D_ENCRYPT(r, l, 10); 
	D_ENCRYPT(l, r, 12); D_ENCRYPT(r, l, 14); D_ENCRYPT(l, r, 16); D_ENCRYPT(r, l, 18); D_ENCRYPT(l, r, 20); D_ENCRYPT(r, l, 22); 
	D_ENCRYPT(l, r, 24); D_ENCRYPT(r, l, 26); D_ENCRYPT(l, r, 28); D_ENCRYPT(r, l, 30); t = l; l = r; r = t;

        l = ROTATE(l, 3) & 0xffffffffL;
        r = ROTATE(r, 3) & 0xffffffffL;

        PERM_OP(l, r, t,  1, 0x55555555L);
        PERM_OP(r, l, t,  8, 0x00ff00ffL);
        PERM_OP(l, r, t,  2, 0x33333333L);
        PERM_OP(r, l, t, 16, 0x0000ffffL);
        PERM_OP(l, r, t,  4, 0x0f0f0f0fL);
    
        out[0] = r;
        out[1] = l;
  } 

    ll = out[0];
    l2c(ll, b);
    ll = out[1];
    l2c(ll, b);
    y = 0;
    u = 0x80;
    bb[8] = 0;

    /*2*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[2] = cov_2char[c];
    /*3*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[3] = cov_2char[c];
    /*4*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[4] = cov_2char[c];
    /*5*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[5] = cov_2char[c];
    /*6*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[6] = cov_2char[c];
    /*7*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[7] = cov_2char[c];
    /*8*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[8] = cov_2char[c];
    /*9*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[9] = cov_2char[c];
    /*10*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[10] = cov_2char[c];
    /*11*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[11] = cov_2char[c];
    /*12*/ c = 0;
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    c <<= 1; if (bb[y] & u) c |= 1; u >>= 1; if (!u) { y++; u = 0x80; }
    ret[12] = cov_2char[c];
    
    ret[13] = '\0';
}

__device__ binary_semaphore<cuda::thread_scope_device> resultSem(1);

__global__ void crackDes(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize];
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      DES_hash_cuda(dict + (idx * DesCrack::passwordSize ), salt, out);
      if( hash[0] == out[0] && hash[1] == out[1] && hash[2] == out[2] && hash[3] == out[3] && hash[4] == out[4] && 
	      hash[5] == out[5] && hash[6] == out[6] && hash[7] == out[7] && hash[8] == out[8] && hash[9] == out[9] && 
	      hash[10] == out[10] && hash[11] == out[11] && hash[12] == out[12] ){

	              resultSem.acquire();
                  result[0] = *(dict + (idx * DesCrack::passwordSize ) );
                  result[1] = *(dict + (idx * DesCrack::passwordSize ) + 1 );
                  result[2] = *(dict + (idx * DesCrack::passwordSize ) + 2 );
                  result[3] = *(dict + (idx * DesCrack::passwordSize ) + 3 );
                  result[4] = *(dict + (idx * DesCrack::passwordSize ) + 4 );
                  result[5] = *(dict + (idx * DesCrack::passwordSize ) + 5 );
                  result[6] = *(dict + (idx * DesCrack::passwordSize ) + 6 );
                  result[7] = *(dict + (idx * DesCrack::passwordSize ) + 7 );
                  result[8] = *(dict + (idx * DesCrack::passwordSize ) + 8 );
                  __threadfence();
	              resultSem.release();
      }
   }
}

#define CHECK  auto  check { [&]() -> bool{ \
                   DES_hash_cuda(transformed, salt, out); \
                   if( hash[0] == out[0] && hash[1] == out[1] && hash[2] == out[2] && hash[3] == out[3] && hash[4] == out[4] &&  \
	                   hash[5] == out[5] && hash[6] == out[6] && hash[7] == out[7] && hash[8] == out[8] && hash[9] == out[9] &&  \
	                   hash[10] == out[10] && hash[11] == out[11] && hash[12] == out[12] ){ \
 \
	                           resultSem.acquire(); \
                               result[0] = *(transformed  ); \
                               result[1] = *(transformed  + 1 ); \
                               result[2] = *(transformed  + 2 ); \
                               result[3] = *(transformed  + 3 ); \
                               result[4] = *(transformed  + 4 ); \
                               result[5] = *(transformed  + 5 ); \
                               result[6] = *(transformed  + 6 ); \
                               result[7] = *(transformed  + 7 ); \
                               result[8] = *(transformed  + 8 ); \
                               __threadfence(); \
	                           resultSem.release(); \
                               return true; \
                   } \
                   return false; \
               } \
   }

   #define RESETFROM auto resetFrom  { [&](int start, const char* word){ \
                               transformed[0] = word[0]; \
                               transformed[1] = word[1]; \
                               transformed[2] = word[2]; \
                               transformed[3] = word[3]; \
                               transformed[4] = word[4]; \
                               transformed[5] = word[5]; \
                               transformed[6] = word[6]; \
                               transformed[7] = word[7]; \
                   } \
   }

   #define RESETFROMCASETOGGLED auto resetFromCaseToggled  { [&](int start, const char* word){ \
                          for(int idx{start}; idx < DesCrack::passwordSize - 1; idx++) {\
                                if(word[idx] >= 0x61 && word[idx] <= 0x7A ){ \
                                       transformed[idx] = word[idx] - 0x20; \
                                       if(check()) return; \
                                }else if(word[idx] >= 0x41 && word[idx] <= 0x5A ){ \
                                       transformed[idx] = word[idx] + 0x20; \
                                       if(check()) return; \
                                } \
                            } \
                   } \
   }

   #define RESETSHIFT2 auto resetShift2{ [&](const char* word){ \
                               transformed[0] = 0; \
                               transformed[1] = 0; \
                               transformed[2] = word[0]; \
                               transformed[3] = word[1]; \
                               transformed[4] = word[2]; \
                               transformed[5] = word[3]; \
                               transformed[6] = word[4]; \
                               transformed[7] = word[5]; \
                   } \
   }

   #define RESETSHIFT1 auto resetShift1{ [&](const char* word){ \
                               transformed[0] = 0; \
                               transformed[1] = word[0]; \
                               transformed[2] = word[1]; \
                               transformed[3] = word[2]; \
                               transformed[4] = word[3]; \
                               transformed[5] = word[4]; \
                               transformed[6] = word[5]; \
                               transformed[7] = word[6]; \
                   } \
   }

   #define WORDSIZE auto wordSize  { [&](const char* word) -> size_t { \
                          if(word[0] == 0 ) return 0; \
                          if(word[1] == 0 ) return 1; \
                          if(word[2] == 0 ) return 2; \
                          if(word[3] == 0 ) return 3; \
                          if(word[4] == 0 ) return 4; \
                          if(word[5] == 0 ) return 5; \
                          if(word[6] == 0 ) return 6; \
                          if(word[7] == 0 ) return 7; \
                          if(word[8] == 0 ) return 8; \
                          return 0; \
                   } \
   }

   #define RESETFROMREVERSED auto resetFromReversed  { [&](const char* word){ \
                               for(int idx{DesCrack::passwordSize - 1}, didx{0}; idx > -1; idx--) \
                                   if(word[idx] != 0) { \
                                        transformed[didx] = word[idx]; \
                                        didx++; \
                                   } \
                   } \
   }

   #define ADDNUMBERS auto addNumbers { [&](size_t idx1) -> bool { \
                          size_t idx2 = idx1 + 1; \
                          for(int i{0}; i<10 ; i++) \
                             for(int j{0}; j<10 ; j++){ \
                                 transformed[idx1] = 0x30 + i ; \
                                 transformed[idx2] = 0x30 + j ; \
                                 if(check()) return true; \
                             } \
                           if(transformed[0] >= 0x61 && transformed[0] <= 0x7A ){ \
                                 transformed[0] = transformed[0] - 0x20; \
                                 for(int i{0}; i<10 ; i++) \
                                    for(int j{0}; j<10 ; j++){ \
                                        transformed[idx1] = 0x30 + i ; \
                                        transformed[idx2] = 0x30 + j ; \
                                        if(check()) return true; \
                                    } \
                           } \
                           return false; \
                    } \
   }

   #define ADDSINGLENUMBER auto addSingleNumber { [&](size_t idx1) -> bool { \
                          for(int i{0}; i<10 ; i++) { \
                              transformed[idx1] = 0x30 + i ; \
                              if(check()) return true; \
                          } \
                          if(transformed[0] >= 0x61 && transformed[0] <= 0x7A ) {\
                                transformed[0] = transformed[0] - 0x20; \
                                for(int i{0}; i<10 ; i++) { \
                                    transformed[idx1] = 0x30 + i ; \
                                    if(check()) return true; \
                                } \
                          } \
                          return false; \
                    } \
   }

   #define ADDSINGLESPECIAL auto addSingleSpecial { [&](size_t idx1) -> bool { \
                          const char spec[] =  {0x20,0x21,0x22,0x23,0x24,0x25,0x26,0x27,0x28, \
                                                0x29,0x2A,0x2B,0x2C,0x2D,0x2E,0x2F,0x3A,0x3B, \
                                                0x3C,0x3D,0x3E,0x3F,0x40,0x5B,0x5C,0x5D,0x5E, \
                                                0x5F,0x60,0x7B,0x7C,0x7D,0x7E}; \
                          transformed[idx1 + 1 ] = 0; \
                          for(int i{0}; i<sizeof(spec) ; i++) {\
                              transformed[idx1] = spec[i] ; \
                              if(check()) return true; \
                          } \
                          if(transformed[0] >= 0x61 && transformed[0] <= 0x7A ){ \
                                transformed[0] = transformed[0] - 0x20; \
                                for(int i{0}; i<sizeof(spec) ; i++) {\
                                    transformed[idx1] = spec[i] ; \
                                    if(check()) return true; \
                                } \
                          } \
                          return false; \
                    } \
   }

   #define ADDFRONTSINGLESPECIAL auto addFrontSingleSpecial { [&]() -> bool { \
                          const char spec[] =  {0x20,0x21,0x22,0x23,0x24,0x25,0x26,0x27,0x28, \
                                                0x29,0x2A,0x2B,0x2C,0x2D,0x2E,0x2F,0x3A,0x3B, \
                                                0x3C,0x3D,0x3E,0x3F,0x40,0x5B,0x5C,0x5D,0x5E, \
                                                0x5F,0x60,0x7B,0x7C,0x7D,0x7E}; \
                          for(int i{0}; i<sizeof(spec) ; i++) { \
                              transformed[0] = spec[i] ; \
                              if(check()) return true; \
                          } \
                          if(transformed[1] >= 0x61 && transformed[1] <= 0x7A ){ \
                                   transformed[1] = transformed[1] - 0x20; \
                                   for(int i{0}; i<sizeof(spec) ; i++) { \
                                       transformed[0] = spec[i] ; \
                                       if(check()) return true; \
                                   } \
                          } \
                          return false; \
                    } \
   }

   #define ADDNUMBERSFRONT auto addNumbersFront { [&]() -> bool { \
                          for(int i{0}; i<10 ; i++) \
                             for(int j{0}; j<10 ; j++){ \
                                 transformed[0] = 0x30 + i ; \
                                 transformed[1] = 0x30 + j ; \
                                 if(check()) return true; \
                             } \
                           if(transformed[2] >= 0x61 && transformed[2] <= 0x7A ) \
                                transformed[2] = transformed[2] - 0x20; \
                           for(int i{0}; i<10 ; i++) \
                             for(int j{0}; j<10 ; j++){ \
                                 transformed[0] = 0x30 + i ; \
                                 transformed[1] = 0x30 + j ; \
                                 if(check()) return true; \
                             } \
                           return false; \
                    } \
   }

   #define LEET2 auto leet2 { [&](char ch) -> char { \
                               switch(ch){ \
                                   case 'A': \
                                   case 'a': \
                                      return '4'; \
                                   case 'E': \
                                   case 'e': \
                                      return '3'; \
                                   case 'I': \
                                   case 'i': \
                                      return '1'; \
                                   case 'O': \
                                   case 'o': \
                                      return '0'; \
                                   case 'S': \
                                   case 's': \
                                      return '5'; \
                                   case 'T': \
                                   case 't': \
                                      return '7'; \
                                   case 'Z': \
                                   case 'z': \
                                      return '2'; \
                                   default: \
                                      return '\0'; \
                               } \
                            } \
    }

   #define LEET auto leet { [&](char ch) -> char { \
                               switch(ch){ \
                                   case 'A': \
                                   case 'a': \
                                      return '4'; \
                                   case 'E': \
                                   case 'e': \
                                      return '3'; \
                                   case 'I': \
                                   case 'i': \
                                      return '1'; \
                                   case 'O': \
                                   case 'o': \
                                      return '0'; \
                                   default: \
                                      return '\0'; \
                               } \
                            } \
    }

   #define ADDSINGLENUMBERFRONT auto addSingleNumberFront { [&]() -> bool { \
                          for(int i{0}; i<10 ; i++) { \
                                 transformed[0] = 0x30 + i ; \
                                 if(check()) return true; \
                           } \
                           if(transformed[1] >= 0x61 && transformed[1] <= 0x7A ) { \
                                transformed[1] = transformed[1] - 0x20; \
                                for(int i{0}; i<10 ; i++) { \
                                      transformed[0] = 0x30 + i ; \
                                      if(check()) return true; \
                                 } \
                           } \
                           return false; \
                    } \
   }

__global__ void crackDesTr1_1(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROM;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      resetFrom(0, word);
      if(word[0] >= 0x61 && word[0] <= 0x7A ){
             transformed[0] = word[0] - 0x20;
             if(check()) return;
      }
    }
}

__global__ void crackDesTr1_2(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROM;
   ADDSINGLENUMBER;
   WORDSIZE;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      size_t           len = wordSize(word);
      resetFrom(0, word);
      switch(len){
              case 1:
              case 2:
              case 3:
              case 4:
              case 5:
              case 6:
              case 7:
                  if(addSingleNumber(len)) return;
              break;
              case 8:
                  if(addSingleNumber(7)) return;
      }
   }
}

__global__ void crackDesTr1_3(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROM;
   ADDNUMBERS;
   WORDSIZE;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      size_t      len  = wordSize(word);
      resetFrom(0, word);
      switch(len){
              case 1:
              case 2:
              case 3:
              case 4:
              case 5:
              case 6:
                  if(addNumbers(len)) return;
              break;
              case 7:
              case 8:
                  if(addNumbers(6)) return;
      }
   }
}

__global__ void crackDesTr1_4(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROM;
   ADDSINGLESPECIAL;
   ADDFRONTSINGLESPECIAL;
   RESETSHIFT1;
   WORDSIZE;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char*      word = dict + (idx * DesCrack::passwordSize );
      size_t           len  = wordSize(word);
      resetFrom(0, word);
      switch(len){
              case 1:
              case 2:
              case 3:
              case 4:
              case 5:
              case 6:
              case 7:
                  if(addSingleSpecial(len)) return;
              break;
              case 8:
                  if(addSingleSpecial(7)) return;
      }
      resetShift1(word);
      if(addFrontSingleSpecial()) return;
   }
}

__global__ void crackDesTr1_5(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROM;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      resetFrom(0, word);
      for(int idx{0}; idx < DesCrack::passwordSize - 1; idx++){
          if(word[idx] >= 0x61 && word[idx] <= 0x7A ){
             transformed[idx] = word[idx] - 0x20;
             if(check()) return;
          }
      }
      resetFrom(0, word);
      if(word[0] >= 0x61 && word[0] <= 0x7A ) transformed[0] = word[0] - 0x20;
      if(word[2] >= 0x61 && word[2] <= 0x7A ) transformed[2] = word[2] - 0x20;
      if(word[4] >= 0x61 && word[4] <= 0x7A ) transformed[4] = word[4] - 0x20;
      if(word[6] >= 0x61 && word[6] <= 0x7A ) transformed[6] = word[6] - 0x20;
      if(check()) return;
      resetFrom(0, word);
      if(word[1] >= 0x61 && word[1] <= 0x7A ) transformed[1] = word[1] - 0x20;
      if(word[3] >= 0x61 && word[3] <= 0x7A ) transformed[3] = word[3] - 0x20;
      if(word[5] >= 0x61 && word[5] <= 0x7A ) transformed[5] = word[5] - 0x20;
      if(word[7] >= 0x61 && word[7] <= 0x7A ) transformed[7] = word[7] - 0x20;
      if(check()) return;
      resetFrom(0, word);
      if(word[0] >= 0x61 && word[0] <= 0x7A ) transformed[0] = word[0] - 0x20;
      if(word[3] >= 0x61 && word[3] <= 0x7A ) transformed[3] = word[3] - 0x20;
      if(word[6] >= 0x61 && word[6] <= 0x7A ) transformed[6] = word[6] - 0x20;
      if(check()) return;
      resetFrom(0, word);
      if(word[0] >= 0x61 && word[0] <= 0x7A ) transformed[0] = word[0] - 0x20;
      if(word[7] >= 0x61 && word[7] <= 0x7A ) transformed[7] = word[7] - 0x20;
      if(check()) return;
    }
}

__global__ void crackDesTr1_6(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETSHIFT1;
   RESETSHIFT2;
   ADDSINGLENUMBERFRONT;
   ADDNUMBERSFRONT;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      resetShift1(word);
      if(addSingleNumberFront()) return;
      resetShift2(word);
      if(addNumbersFront()) return;
   }
}

__global__ void crackDesTr2_1(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROMREVERSED;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      resetFromReversed(word);
      if(check()) return;
   }
}

__global__ void crackDesTr2_2(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROM;
   RESETFROMREVERSED;
   WORDSIZE;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char*      word = dict + (idx * DesCrack::passwordSize );
      resetFromReversed(word);
      if(check()) return;
      size_t           len = wordSize(word);
      resetFrom(0, word);
      switch(len){
              case 1:
                  transformed[1]=transformed[0];
                  if(check()) return;
              break;
              case 2:
                  transformed[2]=transformed[0];
                  transformed[3]=transformed[1];
                  if(check()) return;
                  transformed[2]=transformed[1];
                  transformed[3]=transformed[0];
                  if(check()) return;
              break;
              case 3:
                  transformed[3]=transformed[0];
                  transformed[4]=transformed[1];
                  transformed[5]=transformed[2];
                  if(check()) return;
                  transformed[3]=transformed[2];
                  transformed[4]=transformed[1];
                  transformed[5]=transformed[0];
                  if(check()) return;
              break;
              case 4:
                  transformed[4]=transformed[0];
                  transformed[5]=transformed[1];
                  transformed[6]=transformed[2];
                  transformed[7]=transformed[3];
                  if(check()) return;
                  transformed[4]=transformed[3];
                  transformed[5]=transformed[2];
                  transformed[6]=transformed[1];
                  transformed[7]=transformed[0];
                  if(check()) return;
              break;
              case 5:
                  transformed[5]=transformed[0];
                  transformed[6]=transformed[1];
                  transformed[7]=transformed[2];
                  if(check()) return;
              break;
              case 6:
                  transformed[6]=transformed[0];
                  transformed[7]=transformed[1];
                  if(check()) return;
              break;
              case 7:
                  transformed[7]=transformed[0];
                  if(check()) return;
              break;
              case 8:
              break;
      }
   }
}

__global__ void crackDesTr2_3(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROM;
   LEET;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      resetFrom(0, word);
      for(int idx{0}; idx < ( DesCrack::passwordSize - 1 ) && 
                              transformed[idx] !=0; idx++){
          char newch = leet(transformed[idx]);
          if(newch != 0 ){
             transformed[idx] = newch;
             if(check()) return;
          }
      }
    }
}

__global__ void crackDesTr2_4(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROM;
   LEET2;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      resetFrom(0, word);
      for(int idx{0}; idx < ( DesCrack::passwordSize - 1 ) && 
                              transformed[idx] !=0; idx++){
          char newch = leet2(transformed[idx]);
          if(newch != 0 ){
             transformed[idx] = newch;
             if(check()) return;
          }
      }
    }
}

__global__ void crackDesTr3_1(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROMCASETOGGLED;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      resetFromCaseToggled(0, word);
      if(check()) return;
    }
}

__global__ void crackDesTr3_2(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETSHIFT1;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      resetShift1(word);
      transformed[0] = transformed[1];
      if(check()) return;
      if(transformed[0] >= 0x61 && transformed[0] <= 0x7A ){
             transformed[0] = transformed[0] - 0x20;
             if(check()) return;
             transformed[1] = transformed[1] - 0x20;
             if(check()) return;
      }
    }
}

__global__ void crackDesTr3_3(const char* dict, size_t rows, const char* hash, char* result){
   char salt[DesCrack::saltSize],
        out[DesCrack::hashSize],
        transformed[DesCrack::passwordSize] = {};
   
   salt[0]=hash[0]; salt[1]=hash[1]; salt[2]=0;
   unsigned int cidx { blockIdx.x * blockDim.x + threadIdx.x },
                cblk { blockDim.x * gridDim.x };
   CHECK;
   RESETFROMREVERSED;
   WORDSIZE;

   for(size_t idx{cidx}; idx<rows; idx += cblk){
      if(result[0] != 0 ) return;
      const char* word = dict + (idx * DesCrack::passwordSize );
      resetFromReversed(word);
      if(check()) return;
      size_t           len = wordSize(word);
      switch(len){
              case 1:
                  transformed[0]=word[0];
                  transformed[1]=word[0];
              break;
              case 2:
                  transformed[0]=word[0];
                  transformed[1]=word[0];
                  transformed[2]=word[1];
                  transformed[3]=word[1];
              break;
              case 3:
                  transformed[0]=word[0];
                  transformed[1]=word[0];
                  transformed[2]=word[1];
                  transformed[3]=word[1];
                  transformed[4]=word[2];
                  transformed[5]=word[2];
              break;
              case 4:
              case 5:
              case 6:
              case 7:
              case 8:
                  transformed[0]=word[0];
                  transformed[1]=word[0];
                  transformed[2]=word[1];
                  transformed[3]=word[1];
                  transformed[4]=word[2];
                  transformed[5]=word[2];
                  transformed[6]=word[3];
                  transformed[7]=word[3];
              break;
      }
      if(check()) return;
   }
}

DesCrack::DesCrack(const string hash, bool  tMode)  noexcept 
      : transformMode { tMode }, 
        group1{
                 [&](size_t blks){crackTr1_1(blks);},
                 [&](size_t blks){crackTr1_6(blks);},
                 [&](size_t blks){crackTr1_2(blks);},
                 [&](size_t blks){crackTr1_4(blks);},
                 [&](size_t blks){crackTr1_5(blks);},
                 [&](size_t blks){crackTr1_3(blks);}
              },
        group2{
                 [&](size_t blks){crackTr2_1(blks);},
                 [&](size_t blks){crackTr2_2(blks);},
                 [&](size_t blks){crackTr2_3(blks);}, 
                 [&](size_t blks){crackTr2_4(blks);} 
              },
        group3{
                 [&](size_t blks){crackTr3_1(blks);},
                 [&](size_t blks){crackTr3_2(blks);},
                 [&](size_t blks){crackTr3_3(blks);}
              }

{
    int         cudaDetectedDevices  { 0 };
    hipError_t errorId              { hipGetDeviceCount(&cudaDetectedDevices) };
    if(errorId != hipSuccess) {
	    cerr << "Error probing Cuda devices: " << errorId << " - " << hipGetErrorString(errorId) << "\n";
	    abort();
    }

    if (cudaDetectedDevices == 0) {
	    cerr << "Error: No Cuda device found\n";
	    abort();
    }

    if(hipMallocManaged(&password, passwordSize) != hipSuccess){
	    cerr << "Error: allocating unified memory\n";
	    abort();
    }

    if(hipMallocManaged(&hashTarget, hashSize)  != hipSuccess){
	    cerr << "Error: allocating unified memory\n";
	    abort();
    }

    if(hash.size() != hashSize - 1){
	    cerr << "Error: invalid hash length\n";
	    abort();
    }

    fill_n(hashTarget, hashSize, 0);
    fill_n(password, passwordSize, 0);
    copy_n(hash.c_str(), hashSize-1, hashTarget);
}

DesCrack::~DesCrack(void)  noexcept{
	hipFree(password);
	hipFree(hashTarget);
	hipFree(dict);
}

void DesCrack::crack(size_t blocks) noexcept{
   cout << "Dictionary attack started\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDes<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr1_1(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : initial-capital\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr1_1<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr1_2(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : append single digit, initial-capital + append digit\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr1_2<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr1_3(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : append single special character, initial-capital + append spec. character\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr1_3<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr1_4(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : append special character, initial-capital + special\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr1_4<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr1_5(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : some upper case combinations\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr1_5<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr1_6(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : add digit(s) at begining, initial-capital + digit(s) at beginning\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr1_6<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr2_1(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : reverse\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr2_1<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr2_2(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : specular, repeated\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr2_2<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr2_3(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : leet\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr2_3<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr2_4(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : leet extended\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr2_4<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr3_1(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : toggle upper/lower case \n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr3_1<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr3_2(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : Initial duplicated lower /upper case\n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr3_2<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

void DesCrack::crackTr3_3(size_t blocks) noexcept{
   cout << "Dictionary attack with transformation  : duplicated characters \n";
   int dim = (rows + blocks - 1) / blocks;
   crackDesTr3_3<<<dim, blocks>>>(dict, rows, hashTarget, password);
   hipDeviceSynchronize();
   if(hasResult()) cout << "Password: " << password << '\n';
   else            cout << "No password found.\n";
}

size_t  DesCrack::countDictItems(void)  noexcept{
    if(! is_regular_file(dictFile)){
	    cerr << "Error: dictionary file wrong path or type.\n";
	    abort();
    }

    ifstream dictionary(dictFile);
    size_t   lines{0};
    for( string line; getline( dictionary, line ); lines++ ){}
    return lines;
}

void DesCrack::loadDict(std::string dFile) noexcept{
    dictFile = dFile;
    rows     = countDictItems();
	hipFree(dict);
    if( hipMallocManaged(&dict, rows * cols) != hipSuccess){
	    cerr << "Error: allocating unified memory\n";
	    abort();
    }
    fill_n(dict, rows * cols, 0);
    ifstream dictionary(dictFile);
    size_t   lines{0};
    for( string line; getline( dictionary, line ); lines++ ){
        if(lines > rows - 1){
            cerr << "Error: dict file changed\n";
            abort();
        }

        for(int i{0}; i < (cols - 1) && i < line.size(); i++)
             *(dict + lines * cols + i ) = line[i];
    }

    cout << "Dictionary loaded: " << lines << " elements\n";
}

 bool DesCrack::hasResult(void)  noexcept{
    return password[0] != 0 ? true : false;
 }

 void  DesCrack::execGroups(size_t gr, size_t blocks) noexcept{
     cout << "Dictionary attack with transformation  : from  group 1 to " << gr << " \n";
     if(gr >= 1){
        for(auto member : group1)
            member(blocks);
        cout << "End group 1.\n";
     }
     if(gr >= 2){
        for(auto member : group2)
            member(blocks);
        cout << "End group 2.\n";
     }
     if(gr >= 3){
        for(auto member : group3)
            member(blocks);
        cout << "End group 3.\n";
     }
 }

 void  DesCrack::execGroup(size_t gr, size_t blocks) noexcept{
     cout << "Dictionary attack with transformation  : from  group " << gr << " \n";
     switch(gr){
         case 1:
            for(auto member : group1)
                member(blocks);
            cout << "End group 1.\n";
         break;
         case 2:
            for(auto member : group2)
                member(blocks);
            cout << "End group 2.\n";
         break;
         case 3:
            for(auto member : group3)
                member(blocks);
            cout << "End group 3.\n";
         break;
     }
 }
 
 } // End Namespace